#include "hip/hip_runtime.h"
/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA. Main file.
 *
 * Copyright (C) 2009 Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under
 * the terms of the MIT License. Read the full licence:
 * http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home
 * page in your work.
 *
 */

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#include "hipify_polyfill.h"
#include "comparison_helpers.h"
#include "cuda_helpers.h"
#include "load_data.h"
#include "print_helpers.h"
#include "stats_logger.h"
#include "stdafx.h"

// #include "vlc_kernel_gm32.cu"
// #include "vlc_kernel_sm32.cu"
#include "vlc_kernel_sm64huff.cuh"
// #include "vlc_kernel_dpt.cu"
// #include "vlc_kernel_dptt.cu"
// #include "scan_kernel.cu"
#include "cpuencode.h"
#include "pack_kernels.cuh"

// ========================================================
// #include "scan.cuh"

/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <math.h>

// ========================================================
// #include "cutil.h"
////////////////////////////////////////////////////////////////////////////
//! Macros

#if CUDART_VERSION >= 4000
#define CUT_DEVICE_SYNCHRONIZE() hipDeviceSynchronize();
#else
#define CUT_DEVICE_SYNCHRONIZE() hipDeviceSynchronize();
#endif

#if CUDART_VERSION >= 4000
#define CUT_DEVICE_RESET() hipDeviceReset();
#else
#define CUT_DEVICE_RESET() hipDeviceReset();
#endif

// This is for the CUTIL bank checker
#ifdef _DEBUG
#if __DEVICE_EMULATION__
// Interface for bank conflict checker
#define CUT_BANK_CHECKER(array, index)                                         \
    (cutCheckBankAccess(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x,     \
                        blockDim.y, blockDim.z, __FILE__, __LINE__, #array,    \
                        index),                                                \
     array[index])
#else
#define CUT_BANK_CHECKER(array, index) array[index]
#endif
#else
#define CUT_BANK_CHECKER(array, index) array[index]
#endif

#define CU_SAFE_CALL_NO_SYNC(call)                                             \
    {                                                                          \
        hipError_t err = call;                                                   \
        if (hipSuccess != err) {                                             \
            fprintf(stderr, "Cuda driver error %x in file '%s' in line %i.\n", \
                    err, __FILE__, __LINE__);                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CU_SAFE_CALL(call) CU_SAFE_CALL_NO_SYNC(call);

#define CU_SAFE_CTX_SYNC()                                                     \
    {                                                                          \
        hipError_t err = hipCtxSynchronize();                                     \
        if (hipSuccess != err) {                                             \
            fprintf(stderr, "Cuda driver error %x in file '%s' in line %i.\n", \
                    err, __FILE__, __LINE__);                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CUDA_SAFE_CALL_NO_SYNC(call)                                           \
    {                                                                          \
        hipError_t err = call;                                                  \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                    __FILE__, __LINE__, hipGetErrorString(err));              \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CUDA_SAFE_CALL(call) CUDA_SAFE_CALL_NO_SYNC(call);

#define CUDA_SAFE_THREAD_SYNC()                                                \
    {                                                                          \
        hipError_t err = CUT_DEVICE_SYNCHRONIZE();                              \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                    __FILE__, __LINE__, hipGetErrorString(err));              \
        }                                                                      \
    }

#define CUFFT_SAFE_CALL(call)                                                  \
    {                                                                          \
        hipfftResult err = call;                                                \
        if (HIPFFT_SUCCESS != err) {                                            \
            fprintf(stderr, "CUFFT error in file '%s' in line %i.\n",          \
                    __FILE__, __LINE__);                                       \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CUT_SAFE_CALL(call)                                                    \
    if (CUTTrue != call) {                                                     \
        fprintf(stderr, "Cut error in file '%s' in line %i.\n", __FILE__,      \
                __LINE__);                                                     \
        exit(EXIT_FAILURE);                                                    \
    }

//! Check for CUDA error
#ifdef _DEBUG
#define CUT_CHECK_ERROR(errorMessage)                                          \
    {                                                                          \
        hipError_t err = hipGetLastError();                                  \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
        err = CUT_DEVICE_SYNCHRONIZE();                                        \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }
#else
#define CUT_CHECK_ERROR(errorMessage)                                          \
    {                                                                          \
        hipError_t err = hipGetLastError();                                  \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }
#endif

//! Check for malloc error
#define CUT_SAFE_MALLOC(mallocCall)                                            \
    {                                                                          \
        if (!(mallocCall)) {                                                   \
            fprintf(stderr, "Host malloc failure in file '%s' in line %i\n",   \
                    __FILE__, __LINE__);                                       \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }                                                                          \
    while (0)                                                                  \
        ;

//! Check if conditon is true (flexible assert)
#define CUT_CONDITION(val)                                                     \
    if (CUTFalse == cutCheckCondition(val, __FILE__, __LINE__)) {              \
        exit(EXIT_FAILURE);                                                    \
    }

#if __DEVICE_EMULATION__

#define CUT_DEVICE_INIT(ARGC, ARGV)

#else

#define CUT_DEVICE_INIT(ARGC, ARGV)                                            \
    {                                                                          \
        int deviceCount;                                                       \
        CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));              \
        if (deviceCount == 0) {                                                \
            fprintf(stderr, "cutil error: no devices supporting CUDA.\n");     \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
        int dev = 0;                                                           \
        cutGetCmdLineArgumenti(ARGC, (const char **)ARGV, "device", &dev);     \
        if (dev < 0)                                                           \
            dev = 0;                                                           \
        if (dev > deviceCount - 1)                                             \
            dev = deviceCount - 1;                                             \
        hipDeviceProp_t deviceProp;                                             \
        CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));     \
        if (cutCheckCmdLineFlag(ARGC, (const char **)ARGV, "quiet") ==         \
            CUTFalse)                                                          \
            fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);    \
        CUDA_SAFE_CALL(hipSetDevice(dev));                                    \
    }

//! Check for CUDA context lost
#define CUDA_CHECK_CTX_LOST(errorMessage)                                      \
    {                                                                          \
        hipError_t err = hipGetLastError();                                  \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
        err = CUT_DEVICE_SYNCHRONIZE();                                        \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

//! Check for CUDA context lost
#define CU_CHECK_CTX_LOST(errorMessage)                                        \
    {                                                                          \
        hipError_t err = hipGetLastError();                                  \
        if (hipErrorInvalidContext != err) {                               \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
        err = CUT_DEVICE_SYNCHRONIZE();                                        \
        if (hipSuccess != err) {                                              \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",  \
                    errorMessage, __FILE__, __LINE__,                          \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#endif

#define CUT_DEVICE_INIT_DRV(cuDevice, ARGC, ARGV)                              \
    {                                                                          \
        cuDevice = 0;                                                          \
        int deviceCount = 0;                                                   \
        hipError_t err = hipInit(0);                                              \
        if (hipSuccess == err)                                               \
            CU_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));              \
        if (deviceCount == 0) {                                                \
            fprintf(stderr, "cutil error: no devices supporting CUDA\n");      \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
        int dev = 0;                                                           \
        cutGetCmdLineArgumenti(ARGC, (const char **)ARGV, "device", &dev);     \
        if (dev < 0)                                                           \
            dev = 0;                                                           \
        if (dev > deviceCount - 1)                                             \
            dev = deviceCount - 1;                                             \
        CU_SAFE_CALL_NO_SYNC(hipDeviceGet(&cuDevice, dev));                     \
        char name[100];                                                        \
        hipDeviceGetName(name, 100, cuDevice);                                  \
        if (cutCheckCmdLineFlag(ARGC, (const char **)ARGV, "quiet") ==         \
            CUTFalse)                                                          \
            fprintf(stderr, "Using device %d: %s\n", dev, name);               \
    }

#define CUT_EXIT(argc, argv)                                                   \
    if (!cutCheckCmdLineFlag(argc, (const char **)argv, "noprompt")) {         \
        printf("\nPress ENTER to exit...\n");                                  \
        fflush(stdout);                                                        \
        fflush(stderr);                                                        \
        getchar();                                                             \
    }                                                                          \
    exit(EXIT_SUCCESS);
// ========================================================

#include "scanLargeArray_kernel.cuh"

inline bool isPowerOfTwo(int n) { return ((n & (n - 1)) == 0); }

inline int floorPow2(int n) {
#ifdef WIN32
    // method 2
    return 1 << (int)logb((float)n);
#else
    // method 1
    // float nf = (float)n;
    // return 1 << (((*(int*)&nf) >> 23) - 127);
    int exp;
    frexp((float)n, &exp);
    return 1 << (exp - 1);
#endif
}

#define BLOCK_SIZE 256

static unsigned int **g_scanBlockSums;
static unsigned int g_numEltsAllocated = 0;
static unsigned int g_numLevelsAllocated = 0;

static void preallocBlockSums(unsigned int maxNumElements) {
    assert(g_numEltsAllocated == 0); // shouldn't be called

    g_numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;
    int level = 0;

    do {
        unsigned int numBlocks =
            std::max(1, (int)std::ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
            level++;
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (unsigned int **)malloc(level * sizeof(unsigned int *));
    g_numLevelsAllocated = level;
    numElts = maxNumElements;
    level = 0;

    do {
        unsigned int numBlocks =
            std::max(1, (int)std::ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
            CUDA_SAFE_CALL(hipMalloc((void **)&g_scanBlockSums[level++],
                                      numBlocks * sizeof(unsigned int)));
        numElts = numBlocks;
    } while (numElts > 1);

    CUT_CHECK_ERROR("preallocBlockSums");
}

static void deallocBlockSums() {
    for (unsigned int i = 0; i < g_numLevelsAllocated; i++) {
        hipFree(g_scanBlockSums[i]);
    }

    CUT_CHECK_ERROR("deallocBlockSums");

    free((void **)g_scanBlockSums);

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}

static void prescanArrayRecursive(unsigned int *outArray,
                                  const unsigned int *inArray, int numElements,
                                  int level) {
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks =
        std::max(1, (int)std::ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock =
        numElements - (numBlocks - 1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = std::max(1, (int) (numEltsLastBlock / 2));
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;

    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;

        if (!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);

        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock =
            sizeof(unsigned int) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize =
        sizeof(unsigned int) * (numEltsPerBlock + extraSpace);

#ifdef DEBUG
    if (numBlocks > 1) {
        assert(g_numEltsAllocated >= numElements);
    }
#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3 grid(std::max((int) 1, (int) (numBlocks - np2LastBlock)), 1, 1);
    dim3 threads(numThreads, 1, 1);

    // make sure there are no CUDA errors before we start
    CUT_CHECK_ERROR("prescanArrayRecursive before kernels");

    // execute the scan
    if (numBlocks > 1) {
        prescan<true, false><<<grid, threads, sharedMemSize>>>(
            outArray, inArray, g_scanBlockSums[level], numThreads * 2, 0, 0);
        CUT_CHECK_ERROR("prescanWithBlockSums");
        if (np2LastBlock) {
            prescan<true, true><<<1, numThreadsLastBlock, sharedMemLastBlock>>>(
                outArray, inArray, g_scanBlockSums[level], numEltsLastBlock,
                numBlocks - 1, numElements - numEltsLastBlock);
            CUT_CHECK_ERROR("prescanNP2WithBlockSums");
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we
        // need to take all of the last values of the sub-blocks and scan those.
        // This will give us a new value that must be sdded to each block to
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(g_scanBlockSums[level], g_scanBlockSums[level],
                              numBlocks, level + 1);

        uniformAdd<<<grid, threads>>>(outArray, g_scanBlockSums[level],
                                      numElements - numEltsLastBlock, 0, 0);
        CUT_CHECK_ERROR("uniformAdd");
        if (np2LastBlock) {
            uniformAdd<<<1, numThreadsLastBlock>>>(
                outArray, g_scanBlockSums[level], numEltsLastBlock,
                numBlocks - 1, numElements - numEltsLastBlock);
            CUT_CHECK_ERROR("uniformAdd");
        }
    } else if (isPowerOfTwo(numElements)) {
        prescan<false, false><<<grid, threads, sharedMemSize>>>(
            outArray, inArray, 0, numThreads * 2, 0, 0);
        CUT_CHECK_ERROR("prescan");
    } else {
        prescan<false, true><<<grid, threads, sharedMemSize>>>(
            outArray, inArray, 0, numElements, 0, 0);
        CUT_CHECK_ERROR("prescanNP2");
    }
}

static void prescanArray(unsigned int *outArray, unsigned int *inArray,
                         int numElements) {
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}

#endif // _PRESCAN_CU_
// ========================================================

long long get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000000) + tv.tv_usec;
}
void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks = 1);

extern "C" void cpu_vlc_encode(unsigned int *indata, unsigned int num_elements,
                               unsigned int *outdata, unsigned int *outsize,
                               unsigned int *codewords,
                               unsigned int *codewordlens);

int main(int argc, char *argv[]) {
    if (!InitCUDA()) {
        return 0;
    }
    unsigned int num_block_threads = 256;
    if (argc > 1)
        for (int i = 1; i < argc; i++)
            runVLCTest(argv[i], num_block_threads);
    else {
        runVLCTest(NULL, num_block_threads, 1024);
    }
    CUDA_SAFE_CALL(hipDeviceReset());
    return 0;
}

void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks) {
    printf("CUDA! Starting VLC Tests!\n");
    unsigned int
        num_elements; // uint num_elements = num_blocks * num_block_threads;
    unsigned int mem_size; // uint mem_size = num_elements * sizeof(int);
    unsigned int symbol_type_size = sizeof(int);
    //////// LOAD DATA ///////////////
    double H; // entropy
    initParams(file_name, num_block_threads, num_blocks, num_elements, mem_size,
               symbol_type_size);
    printf("Parameters: num_elements: %d, num_blocks: %d, num_block_threads: "
           "%d\n----------------------------\n",
           num_elements, num_blocks, num_block_threads);
    ////////LOAD DATA ///////////////
    uint *sourceData = (uint *)malloc(mem_size);
    uint *destData = (uint *)malloc(mem_size);
    uint *crefData = (uint *)malloc(mem_size);

    uint *codewords = (uint *)malloc(NUM_SYMBOLS * symbol_type_size);
    uint *codewordlens = (uint *)malloc(NUM_SYMBOLS * symbol_type_size);

    uint *cw32 = (uint *)malloc(mem_size);
    uint *cw32len = (uint *)malloc(mem_size);
    uint *cw32idx = (uint *)malloc(mem_size);

    uint *cindex2 = (uint *)malloc(num_blocks * sizeof(int));

    memset(sourceData, 0, mem_size);
    memset(destData, 0, mem_size);
    memset(crefData, 0, mem_size);
    memset(cw32, 0, mem_size);
    memset(cw32len, 0, mem_size);
    memset(cw32idx, 0, mem_size);
    memset(codewords, 0, NUM_SYMBOLS * symbol_type_size);
    memset(codewordlens, 0, NUM_SYMBOLS * symbol_type_size);
    memset(cindex2, 0, num_blocks * sizeof(int));
    //////// LOAD DATA ///////////////
    loadData(file_name, sourceData, codewords, codewordlens, num_elements,
             mem_size, H);

    //////// LOAD DATA ///////////////

    unsigned int *d_sourceData, *d_destData, *d_destDataPacked;
    unsigned int *d_codewords, *d_codewordlens;
    unsigned int *d_cw32, *d_cw32len, *d_cw32idx, *d_cindex, *d_cindex2;

    CUDA_SAFE_CALL(hipMalloc((void **)&d_sourceData, mem_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_destData, mem_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_destDataPacked, mem_size));

    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_codewords, NUM_SYMBOLS * symbol_type_size));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_codewordlens, NUM_SYMBOLS * symbol_type_size));

    CUDA_SAFE_CALL(hipMalloc((void **)&d_cw32, mem_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_cw32len, mem_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_cw32idx, mem_size));

    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_cindex, num_blocks * sizeof(unsigned int)));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_cindex2, num_blocks * sizeof(unsigned int)));

    CUDA_SAFE_CALL(
        hipMemcpy(d_sourceData, sourceData, mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_codewords, codewords,
                              NUM_SYMBOLS * symbol_type_size,
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_codewordlens, codewordlens,
                              NUM_SYMBOLS * symbol_type_size,
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(
        hipMemcpy(d_destData, destData, mem_size, hipMemcpyHostToDevice));

    dim3 grid_size(num_blocks, 1, 1);
    dim3 block_size(num_block_threads, 1, 1);
    unsigned int sm_size;

    unsigned int NT = 10; // number of runs for each execution time

    //////////////////* CPU ENCODER *///////////////////////////////////
    unsigned int refbytesize;
    long long timer = get_time();
    cpu_vlc_encode((unsigned int *)sourceData, num_elements,
                   (unsigned int *)crefData, &refbytesize, codewords,
                   codewordlens);
    float msec = (float)((get_time() - timer) / 1000.0);
    printf("CPU Encoding time (CPU): %f (ms)\n", msec);
    printf("CPU Encoded to %d [B]\n", refbytesize);
    unsigned int num_ints = refbytesize / 4 + ((refbytesize % 4 == 0) ? 0 : 1);
    //////////////////* END CPU *///////////////////////////////////

    //////////////////* SM64HUFF KERNEL *///////////////////////////////////
    grid_size.x = num_blocks;
    block_size.x = num_block_threads;
    sm_size = block_size.x * sizeof(unsigned int);
#ifdef CACHECWLUT
    sm_size =
        2 * NUM_SYMBOLS * sizeof(int) + block_size.x * sizeof(unsigned int);
#endif
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for (int i = 0; i < NT; i++) {
        vlc_encode_kernel_sm64huff<<<grid_size, block_size, sm_size>>>(
            d_sourceData, d_codewords, d_codewordlens,
#ifdef TESTING
            d_cw32, d_cw32len, d_cw32idx,
#endif
            d_destData, d_cindex); // testedOK2
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    CUT_CHECK_ERROR("Kernel execution failed\n");
    printf("GPU Encoding time (SM64HUFF): %f (ms)\n", elapsedTime / NT);
    //////////////////* END KERNEL *///////////////////////////////////

#ifdef TESTING
    unsigned int num_scan_elements = grid_size.x;
    preallocBlockSums(num_scan_elements);
    hipMemset(d_destDataPacked, 0, mem_size);
    printf("Num_blocks to be passed to scan is %d.\n", num_scan_elements);
    prescanArray(d_cindex2, d_cindex, num_scan_elements);

    pack2<<<num_scan_elements / 16, 16>>>(
        (unsigned int *)d_destData, d_cindex, d_cindex2,
        (unsigned int *)d_destDataPacked, num_elements / num_scan_elements);
    CUT_CHECK_ERROR("Pack2 Kernel execution failed\n");
    deallocBlockSums();

    CUDA_SAFE_CALL(hipMemcpy(destData, d_destDataPacked, mem_size,
                              hipMemcpyDeviceToHost));
    compare_vectors((unsigned int *)crefData, (unsigned int *)destData,
                    num_ints);
#endif

    free(sourceData);
    free(destData);
    free(codewords);
    free(codewordlens);
    free(cw32);
    free(cw32len);
    free(crefData);
    CUDA_SAFE_CALL(hipFree(d_sourceData));
    CUDA_SAFE_CALL(hipFree(d_destData));
    CUDA_SAFE_CALL(hipFree(d_destDataPacked));
    CUDA_SAFE_CALL(hipFree(d_codewords));
    CUDA_SAFE_CALL(hipFree(d_codewordlens));
    CUDA_SAFE_CALL(hipFree(d_cw32));
    CUDA_SAFE_CALL(hipFree(d_cw32len));
    CUDA_SAFE_CALL(hipFree(d_cw32idx));
    CUDA_SAFE_CALL(hipFree(d_cindex));
    CUDA_SAFE_CALL(hipFree(d_cindex2));
    free(cindex2);
}
