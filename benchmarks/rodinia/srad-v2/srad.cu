#include "hip/hip_runtime.h"
// includes, system
#include "srad.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
__global__ void srad_cuda_1(float *E_C, float *W_C, float *N_C, float *S_C,
                            float *J_cuda, float *C_cuda, int cols, int rows,
                            float q0sqr) {

    // block id
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // thread id
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // indices
    int index = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
    int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
    int index_s =
        cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
    int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
    int index_e =
        cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

    float n, w, e, s, jc, g2, l, num, den, qsqr, c;

    // shared memory allocation
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float east[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float west[BLOCK_SIZE][BLOCK_SIZE];

    // load data to shared memory
    north[ty][tx] = J_cuda[index_n];
    south[ty][tx] = J_cuda[index_s];
    if (by == 0) {
        north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx];
    } else if (by == gridDim.y - 1) {
        south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) +
                               BLOCK_SIZE * bx + cols * (BLOCK_SIZE - 1) + tx];
    }
    __syncthreads();

    west[ty][tx] = J_cuda[index_w];
    east[ty][tx] = J_cuda[index_e];

    if (bx == 0) {
        west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty];
    } else if (bx == gridDim.x - 1) {
        east[ty][tx] =
            J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * (gridDim.x - 1) +
                   cols * ty + BLOCK_SIZE - 1];
    }

    __syncthreads();

    temp[ty][tx] = J_cuda[index];

    __syncthreads();

    jc = temp[ty][tx];

    if (ty == 0 && tx == 0) { // nw
        n = north[ty][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = west[ty][tx] - jc;
        e = temp[ty][tx + 1] - jc;
    } else if (ty == 0 && tx == BLOCK_SIZE - 1) { // ne
        n = north[ty][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = east[ty][tx] - jc;
    } else if (ty == BLOCK_SIZE - 1 && tx == BLOCK_SIZE - 1) { // se
        n = temp[ty - 1][tx] - jc;
        s = south[ty][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = east[ty][tx] - jc;
    } else if (ty == BLOCK_SIZE - 1 && tx == 0) { // sw
        n = temp[ty - 1][tx] - jc;
        s = south[ty][tx] - jc;
        w = west[ty][tx] - jc;
        e = temp[ty][tx + 1] - jc;
    }

    else if (ty == 0) { // n
        n = north[ty][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = temp[ty][tx + 1] - jc;
    } else if (tx == BLOCK_SIZE - 1) { // e
        n = temp[ty - 1][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = east[ty][tx] - jc;
    } else if (ty == BLOCK_SIZE - 1) { // s
        n = temp[ty - 1][tx] - jc;
        s = south[ty][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = temp[ty][tx + 1] - jc;
    } else if (tx == 0) { // w
        n = temp[ty - 1][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = west[ty][tx] - jc;
        e = temp[ty][tx + 1] - jc;
    } else { // the data elements which are not on the borders
        n = temp[ty - 1][tx] - jc;
        s = temp[ty + 1][tx] - jc;
        w = temp[ty][tx - 1] - jc;
        e = temp[ty][tx + 1] - jc;
    }

    g2 = (n * n + s * s + w * w + e * e) / (jc * jc);

    l = (n + s + w + e) / jc;

    num = (0.5 * g2) - ((1.0 / 16.0) * (l * l));
    den = 1 + (.25 * l);
    qsqr = num / (den * den);

    // diffusion coefficent (equ 33)
    den = (qsqr - q0sqr) / (q0sqr * (1 + q0sqr));
    c = 1.0 / (1.0 + den);

    // saturate diffusion coefficent
    if (c < 0) {
        temp_result[ty][tx] = 0;
    } else if (c > 1) {
        temp_result[ty][tx] = 1;
    } else {
        temp_result[ty][tx] = c;
    }

    __syncthreads();

    C_cuda[index] = temp_result[ty][tx];
    E_C[index] = e;
    W_C[index] = w;
    S_C[index] = s;
    N_C[index] = n;
}

__global__ void srad_cuda_2(float *E_C, float *W_C, float *N_C, float *S_C,
                            float *J_cuda, float *C_cuda, int cols, int rows,
                            float lambda, float q0sqr) {
    // block id
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // thread id
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // indices
    int index = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
    int index_s =
        cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
    int index_e =
        cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
    float cc, cn, cs, ce, cw, d_sum;

    // shared memory allocation
    __shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float east_c[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];

    // load data to shared memory
    temp[ty][tx] = J_cuda[index];

    __syncthreads();

    south_c[ty][tx] = C_cuda[index_s];

    if (by == gridDim.y - 1) {
        south_c[ty][tx] =
            C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx +
                   cols * (BLOCK_SIZE - 1) + tx];
    }
    __syncthreads();

    east_c[ty][tx] = C_cuda[index_e];

    if (bx == gridDim.x - 1) {
        east_c[ty][tx] =
            C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * (gridDim.x - 1) +
                   cols * ty + BLOCK_SIZE - 1];
    }

    __syncthreads();

    c_cuda_temp[ty][tx] = C_cuda[index];

    __syncthreads();

    cc = c_cuda_temp[ty][tx];

    if (ty == BLOCK_SIZE - 1 && tx == BLOCK_SIZE - 1) { // se
        cn = cc;
        cs = south_c[ty][tx];
        cw = cc;
        ce = east_c[ty][tx];
    } else if (tx == BLOCK_SIZE - 1) { // e
        cn = cc;
        cs = c_cuda_temp[ty + 1][tx];
        cw = cc;
        ce = east_c[ty][tx];
    } else if (ty == BLOCK_SIZE - 1) { // s
        cn = cc;
        cs = south_c[ty][tx];
        cw = cc;
        ce = c_cuda_temp[ty][tx + 1];
    } else { // the data elements which are not on the borders
        cn = cc;
        cs = c_cuda_temp[ty + 1][tx];
        cw = cc;
        ce = c_cuda_temp[ty][tx + 1];
    }

    // divergence (equ 58)
    d_sum =
        cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];

    // image update (equ 61)
    c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;

    __syncthreads();

    J_cuda[index] = c_cuda_result[ty][tx];
}

void random_matrix(float *I, int rows, int cols);
void runTest(int argc, char **argv);
void usage(int argc, char **argv) {
    fprintf(
        stderr,
        "Usage: %s <rows> <cols> <y1> <y2> <x1> <x2> <lamda> <no. of iter>\n",
        argv[0]);
    fprintf(stderr, "\t<rows>   - number of rows\n");
    fprintf(stderr, "\t<cols>    - number of cols\n");
    fprintf(stderr, "\t<y1> 	 - y1 value of the speckle\n");
    fprintf(stderr, "\t<y2>      - y2 value of the speckle\n");
    fprintf(stderr, "\t<x1>       - x1 value of the speckle\n");
    fprintf(stderr, "\t<x2>       - x2 value of the speckle\n");
    fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
    fprintf(stderr, "\t<no. of iter>   - number of iterations\n");

    exit(1);
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
    runTest(argc, argv);

    return EXIT_SUCCESS;
}

void runTest(int argc, char **argv) {
    int rows, cols, size_I, size_R, niter = 10, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;

#ifdef CPU
    float Jc, G2, L, num, den, qsqr;
    int *iN, *iS, *jE, *jW, k;
    float *dN, *dS, *dW, *dE;
    float cN, cS, cW, cE, D;
#endif

#ifdef GPU

    float *J_cuda;
    float *C_cuda;
    float *E_C, *W_C, *N_C, *S_C;

#endif

    unsigned int r1, r2, c1, c2;
    float *c;

    if (argc == 9) {
        rows = atoi(argv[1]); // number of rows in the domain
        cols = atoi(argv[2]); // number of cols in the domain
        if ((rows % 16 != 0) || (cols % 16 != 0)) {
            fprintf(stderr, "rows and cols must be multiples of 16\n");
            exit(1);
        }
        r1 = atoi(argv[3]);     // y1 position of the speckle
        r2 = atoi(argv[4]);     // y2 position of the speckle
        c1 = atoi(argv[5]);     // x1 position of the speckle
        c2 = atoi(argv[6]);     // x2 position of the speckle
        lambda = atof(argv[7]); // Lambda value
        niter = atoi(argv[8]);  // number of iterations

    } else {
        usage(argc, argv);
    }

    size_I = cols * rows;
    size_R = (r2 - r1 + 1) * (c2 - c1 + 1);

    I = (float *)malloc(size_I * sizeof(float));
    J = (float *)malloc(size_I * sizeof(float));
    c = (float *)malloc(sizeof(float) * size_I);

#ifdef CPU

    iN = (int *)malloc(sizeof(unsigned int *) * rows);
    iS = (int *)malloc(sizeof(unsigned int *) * rows);
    jW = (int *)malloc(sizeof(unsigned int *) * cols);
    jE = (int *)malloc(sizeof(unsigned int *) * cols);

    dN = (float *)malloc(sizeof(float) * size_I);
    dS = (float *)malloc(sizeof(float) * size_I);
    dW = (float *)malloc(sizeof(float) * size_I);
    dE = (float *)malloc(sizeof(float) * size_I);

    for (int i = 0; i < rows; i++) {
        iN[i] = i - 1;
        iS[i] = i + 1;
    }
    for (int j = 0; j < cols; j++) {
        jW[j] = j - 1;
        jE[j] = j + 1;
    }
    iN[0] = 0;
    iS[rows - 1] = rows - 1;
    jW[0] = 0;
    jE[cols - 1] = cols - 1;

#endif

#ifdef GPU

    // Allocate device memory
    hipMalloc((void **)&J_cuda, sizeof(float) * size_I);
    hipMalloc((void **)&C_cuda, sizeof(float) * size_I);
    hipMalloc((void **)&E_C, sizeof(float) * size_I);
    hipMalloc((void **)&W_C, sizeof(float) * size_I);
    hipMalloc((void **)&S_C, sizeof(float) * size_I);
    hipMalloc((void **)&N_C, sizeof(float) * size_I);

#endif

    printf("Randomizing the input matrix\n");
    // Generate a random matrix
    random_matrix(I, rows, cols);

    for (int k = 0; k < size_I; k++) {
        J[k] = (float)exp(I[k]);
    }
    printf("Start the SRAD main loop\n");
    for (iter = 0; iter < niter; iter++) {
        sum = 0;
        sum2 = 0;
        for (int i = r1; i <= r2; i++) {
            for (int j = c1; j <= c2; j++) {
                tmp = J[i * cols + j];
                sum += tmp;
                sum2 += tmp * tmp;
            }
        }
        meanROI = sum / size_R;
        varROI = (sum2 / size_R) - meanROI * meanROI;
        q0sqr = varROI / (meanROI * meanROI);

#ifdef CPU

        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {

                k = i * cols + j;
                Jc = J[k];

                // directional derivates
                dN[k] = J[iN[i] * cols + j] - Jc;
                dS[k] = J[iS[i] * cols + j] - Jc;
                dW[k] = J[i * cols + jW[j]] - Jc;
                dE[k] = J[i * cols + jE[j]] - Jc;

                G2 = (dN[k] * dN[k] + dS[k] * dS[k] + dW[k] * dW[k] +
                      dE[k] * dE[k]) /
                     (Jc * Jc);

                L = (dN[k] + dS[k] + dW[k] + dE[k]) / Jc;

                num = (0.5 * G2) - ((1.0 / 16.0) * (L * L));
                den = 1 + (.25 * L);
                qsqr = num / (den * den);

                // diffusion coefficent (equ 33)
                den = (qsqr - q0sqr) / (q0sqr * (1 + q0sqr));
                c[k] = 1.0 / (1.0 + den);

                // saturate diffusion coefficent
                if (c[k] < 0) {
                    c[k] = 0;
                } else if (c[k] > 1) {
                    c[k] = 1;
                }
            }
        }
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {

                // current index
                k = i * cols + j;

                // diffusion coefficent
                cN = c[k];
                cS = c[iS[i] * cols + j];
                cW = c[k];
                cE = c[i * cols + jE[j]];

                // divergence (equ 58)
                D = cN * dN[k] + cS * dS[k] + cW * dW[k] + cE * dE[k];

                // image update (equ 61)
                J[k] = J[k] + 0.25 * lambda * D;
            }
        }

#endif // CPU

#ifdef GPU

        // Currently the input size must be divided by 16 - the block size
        int block_x = cols / BLOCK_SIZE;
        int block_y = rows / BLOCK_SIZE;

        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(block_x, block_y);

        // Copy data from main memory to device memory
        hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice);

        // Run kernels
        srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,
                                           cols, rows, q0sqr);
        srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,
                                           cols, rows, lambda, q0sqr);

        // Copy data from device memory to main memory
        hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost);

#endif
    }

    hipDeviceSynchronize();

#ifdef OUTPUT
    // Printing output
    printf("Printing Output:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.5f ", J[i * cols + j]);
        }
        printf("\n");
    }
#endif

    printf("Computation Done\n");

    free(I);
    free(J);
#ifdef CPU
    free(iN);
    free(iS);
    free(jW);
    free(jE);
    free(dN);
    free(dS);
    free(dW);
    free(dE);
#endif
#ifdef GPU
    hipFree(C_cuda);
    hipFree(J_cuda);
    hipFree(E_C);
    hipFree(W_C);
    hipFree(N_C);
    hipFree(S_C);
#endif
    free(c);
}

void random_matrix(float *I, int rows, int cols) {

    srand(7);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            I[i * cols + j] = rand() / (float)RAND_MAX;
        }
    }
}
