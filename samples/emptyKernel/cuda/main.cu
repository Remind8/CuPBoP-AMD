#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void empty(int* N) {
  *N = 10;
}

int main(int argc, char *argv[]) {

  printf("Testing main");
  int device;
  hipDeviceProp_t prop;

  hipGetDevice(&device); // test runtime mapped

  hipGetDeviceProperties(&prop, device);
  int n = 1;

  int *h_a;

  int *d_a;

  size_t bytes = n*sizeof(int);

  h_a = (int*)malloc(bytes);

  hipMalloc(&d_a, bytes);


  for(int i = 0; i < n; i++ ) {
    h_a[i] = 0;
  }

  hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);

  int blockSize, gridSize;

  blockSize = 1024;
  gridSize = (int)ceil((float)n/blockSize);


  empty<<<gridSize, blockSize>>>(d_a);

  hipMemcpy( h_a, d_a, bytes, hipMemcpyDeviceToHost );

  // check the value

  // h_a = 0
  if (h_a[0] == 10) {
    printf("correct \n");
  }


  hipFree(d_a);
  free(h_a);

  return 0;

}