#include "hip/hip_runtime.h"
#include "stdio.h"
__global__ void empty(int* N) {
  *N = 10;
}

int main(int argc, char **argv) { 
  int * host_n;
  int * device_n;

  host_n = (int*)malloc(sizeof(int));
  hipMalloc(&device_n, sizeof(int));

  empty<<<1,1>>>(device_n);

  hipMemcpy(host_n, device_n, sizeof(int), hipMemcpyDeviceToHost );

  if(*host_n == 10) {
    printf("Success \n");
  }

  hipFree(device_n);
  return 0;
}
