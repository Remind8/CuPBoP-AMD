#include "hip/hip_runtime.h"
#include "stdio.h"

// __global__ void test1(int* A, unsigned int* B, unsigned long long int* C)
// {
//   atomicOr(A,10); // 1010

//   atomicOr(B,7);  // 0111

//   atomicOr(C,5);  // 0101
// }


// __global__ void test2(int* A, unsigned int* B, unsigned long long int* C, float* D)
// {
// 	atomicAdd(A,10);

// 	atomicAdd(B,10);

// 	atomicAdd(C,10);

// 	atomicAdd(D,10.8);

// }

// __global__ void test3 (int* A, unsigned int* B, unsigned long long int* C)
// {
//   atomicAnd(A,10);

//   atomicAnd(B,1);

//   atomicAnd(C,5);
// }


// __global__ void test4 (int* A, unsigned int* B, unsigned long long int* C)
// {
//   atomicCAS(A,2,11);

//   atomicCAS(B,5,1);

//   atomicCAS(C,7,3);
// }

// __global__ void test5(int* A, unsigned int* B, unsigned long long int* C, float* D)
// {
// 	atomicExch(A,10);

// 	atomicExch(B,100);

// 	atomicExch(C,20);

// 	atomicExch(D,200.0);
// }

// __global__ void test6(unsigned int* B)
// {
//   atomicDec(B,7);//0111 -> 1000 -> 0000 -> 0001 -> 0010 -> 0011 -> 0100 -> 0101 -> 0110 ...
//   	  /*the second argument on atomicDec() is a limit for decs. When this limit is reached, B receives <LIM>*/
// }

// __global__ void test7(unsigned int* B)
// {
//   atomicInc(B,7);//0111 -> 1000 -> 0000 -> 0001 -> 0010 -> 0011 -> 0100 -> 0101 -> 0110 ...
//   	  /*the second argument on atomicInc() is a limit for increments. When this limit is reached, B receives 0*/
// }

// __global__ void test8(int* A, unsigned int* B, unsigned long long int* C)
// {
//   atomicMax(A,10);

//   atomicMax(B,1);

//   atomicMax(C,5);
// }

// __global__ void test9(int* A, unsigned int* B, unsigned long long int* C)
// {
//   atomicMin(A,10);

//   atomicMin(B,1);

//   atomicMin(C,5);
// }

// __global__ void test10(int* A, unsigned int* B)
// {
// 	atomicSub(A,10);

// 	atomicSub(B,5);

// }

__global__ void test11(int* A, unsigned int* B, unsigned long long int* C)
{
  atomicXor(A,10);//1010 xor 0101 = 1111 /*xor looks for distinct bits*/

  atomicXor(B,7);//0111 xor 0101 = 0010

  atomicXor(C,5);//0101 xor 0101 = 0000
}



int main() {

  int NUM_ELEMENTS = 1;
  int* host_a; 
  unsigned int* host_b;
  unsigned long long int* host_c;
  float* host_d;

  int* device_a;
  unsigned int* device_b;
  unsigned long long int* device_c;
  float* device_d;

  hipMalloc(&device_a, NUM_ELEMENTS * sizeof(int));

  hipMalloc(&device_b, NUM_ELEMENTS * sizeof(unsigned int));

  hipMalloc(&device_c, NUM_ELEMENTS * sizeof(unsigned long long int));

  hipMalloc(&device_d, NUM_ELEMENTS * sizeof(float));


  host_a = (int*)malloc(NUM_ELEMENTS * sizeof(int));
  host_b = (unsigned int*)malloc(NUM_ELEMENTS * sizeof(unsigned int));
  host_c = (unsigned long long int*)malloc(NUM_ELEMENTS * sizeof(unsigned long long int));
  host_d = (float*)malloc(NUM_ELEMENTS * sizeof(float));
  
  // test 1, test 2, test 5

  *host_a = 0;
  *host_b = 0;
  *host_c = 0;
  *host_d = 0;

  // test 3, test 4

  *host_a = 5; // 2 and 0110
  *host_b = 5; // 0000 and 0001
  *host_c = 5; // 0 and 0101
  
  hipMemcpy(device_a, host_a ,NUM_ELEMENTS * sizeof(int)  , hipMemcpyDeviceToHost);
  hipMemcpy(device_b, host_b ,NUM_ELEMENTS * sizeof(unsigned int)  , hipMemcpyDeviceToHost);
  hipMemcpy(device_c, host_c ,NUM_ELEMENTS * sizeof(unsigned long long int)  , hipMemcpyDeviceToHost);
  hipMemcpy(device_d, host_d ,NUM_ELEMENTS * sizeof(float)  , hipMemcpyDeviceToHost);

  test11<<<1,1>>>(device_a, device_b, device_c);
  // test5<<<1,1>>>(device_a, device_b, device_c, device_d);
  // test4<<<1,1>>>(device_a, device_b, device_c);
  // test7<<<1,1>>>(device_b);
  // test10<<<1,1>>>(device_a, device_b);


  hipMemcpy(host_a, device_a ,NUM_ELEMENTS * sizeof(int)  , hipMemcpyHostToDevice);
  hipMemcpy(host_b, device_b ,NUM_ELEMENTS * sizeof(unsigned int)  , hipMemcpyHostToDevice);
  hipMemcpy(host_c, device_c ,NUM_ELEMENTS * sizeof(unsigned long long int)  , hipMemcpyHostToDevice);
  hipMemcpy(host_d, device_d ,NUM_ELEMENTS * sizeof(float)  , hipMemcpyHostToDevice);


  printf(" %d ", *host_a);
  printf(" %d ", *host_b);
  printf(" %llu \n", *host_c);
  printf(" %f \n", *host_d);


  // check Answers

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  hipFree(device_d);

  free(host_a);
  free(host_b);
  free(host_c);
  free(host_d);


}