#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "stdio.h"

// https://github.com/ricsonc/linear_vs_texture_memory_cuda/blob/master/main.cu

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef float4 typ;

texture<typ, 1> texref;
texture<typ, 2> texref2;
texture<typ, 3> texref3;

/* 1D texture memory */

__global__ void touch1Dtexture(void* outPtr_, long M){
    long N = gridDim.x * blockDim.x;
    long i = blockIdx.x * blockDim.x + threadIdx.x;

    typ* outPtr = (typ*) outPtr_;

    for(; i < M-2; i += N) {
        outPtr[i] = (
            tex1Dfetch(texref, i) +
            tex1Dfetch(texref, i+1) +
            tex1Dfetch(texref, i+2) + 
            tex1Dfetch(texref, i+3)
        );
    }
}

void time1Dtexture()
{
    void* refPtr;
    void* outPtr;
    long M = 1000L*1000L*100L;
    int blocks = 65536;
    int threads = 64;

    gpuCheck( hipMalloc(&refPtr, M*sizeof(typ)) );
    gpuCheck( hipMalloc(&outPtr, M*sizeof(typ)) );
    gpuCheck( hipBindTexture(NULL, texref, refPtr, M) );
    
    touch1Dtexture<<<blocks, threads>>>(outPtr, M);
    
    gpuCheck( hipPeekAtLastError() );
    gpuCheck( hipDeviceSynchronize() );


    gpuCheck( hipUnbindTexture(texref) );    
    gpuCheck( hipFree(refPtr) );
    gpuCheck( hipFree(outPtr) );
    
}

/* 2D linear memory */

__global__ void touch2Dtexture(void* outPtr_, long M){
    long N = gridDim.x * blockDim.x;
    long ix = blockIdx.x * blockDim.x + threadIdx.x;
    long iy = blockIdx.y * blockDim.y + threadIdx.y;

    typ* outPtr = (typ*) outPtr_;

    for(; ix < M-1; ix += N) {
        for(; iy < M-1; iy += N) {
            outPtr[ix*M+iy] = (
                tex2D(texref2, ix, iy) +
                tex2D(texref2, ix+1, iy) +
                tex2D(texref2, ix, iy+1) +
                tex2D(texref2, ix+1, iy+1)
            );
        }
    }
}

/* 3D texture memory */

__global__ void touch3Dtexture(void* outPtr_, long M){
    long N = gridDim.x * blockDim.x;
    long ix = blockIdx.x * blockDim.x + threadIdx.x;
    long iy = blockIdx.y * blockDim.y + threadIdx.y;
    long iz = blockIdx.z * blockDim.z + threadIdx.z;

    typ* outPtr = (typ*) outPtr_;

    for(; ix < M-1; ix += N) {
        for(; iy < M-1; iy += N) {
            for(; iz < M-1; iz += N) {
                outPtr[ix*M*M+iy*M+iz] = (
                    tex3D(texref3, ix, iy, iz) +                    
                    tex3D(texref3, ix, iy, iz+1) +
                    tex3D(texref3, ix, iy+1, iz) +
                    tex3D(texref3, ix+1, iy, iz)
                );
            }
        }
    }
}

void time2Dtexture()
{
    long M = 10000;
    dim3 blocks(256, 256);
    dim3 threads(8,8);

    void* outPtr;
    gpuCheck( hipMalloc(&outPtr, M*M*sizeof(typ)) );

    hipArray *refPtr;
    gpuCheck( hipMallocArray(&refPtr, &texref2.channelDesc, M, M) );
    gpuCheck( hipBindTextureToArray(texref2, refPtr) );
    
    auto start = std::chrono::system_clock::now();
    for(int i = 0; i < RUNS; i++){
        touch2Dtexture<<<blocks, threads>>>(outPtr, M);
    }
    
    gpuCheck( hipPeekAtLastError() );
    gpuCheck( hipDeviceSynchronize() );

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> delta = end-start;

    gpuCheck( hipUnbindTexture(texref) );    
    gpuCheck( hipFreeArray(refPtr) );
    gpuCheck( hipFree(outPtr) );
    
    printf("texture 2D: %.1f ms\n", delta.count());
}


void time3Dtexture()
{
    unsigned long M = 465;
    dim3 blocks(32,32,32);
    dim3 threads(4,4,4);

    void* outPtr;
    gpuCheck( hipMalloc(&outPtr, M*M*M*sizeof(typ)) );
    
    hipArray* refPtr;
    gpuCheck( hipMalloc3DArray(&refPtr, &texref2.channelDesc, {M, M, M}) );
    gpuCheck( hipBindTextureToArray(texref3, refPtr) );
    
    auto start = std::chrono::system_clock::now();
    for(int i = 0; i < RUNS; i++){
        touch3Dtexture<<<blocks, threads>>>(outPtr, M);
    }
    
    gpuCheck( hipPeekAtLastError() );
    gpuCheck( hipDeviceSynchronize() );

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> delta = end-start;

    gpuCheck( hipUnbindTexture(texref) );    
    gpuCheck( hipFreeArray(refPtr) );
    gpuCheck( hipFree(outPtr) );
    
    printf("texture 3D: %.1f ms\n", delta.count());
}