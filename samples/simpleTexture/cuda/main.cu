#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

// From: https://forums.developer.nvidia.com/t/simplest-texture-2d-examples/71823/11

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)
// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

texture<int, 2, hipReadModeElementType> tex;

__global__ void kernel (int m, int n) 
{
    int val;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            val = tex2D (tex, col+0.5f, row+0.5f);
            // printf ("%3d  ", val);
        }
        // printf ("\n");
    }
}

int main (void)
{
    int m = 4; // height = #rows
    int n = 3; // width  = #columns
    size_t pitch, tex_ofs;
    int arr[4][3]= {{10, 11, 12},
                    {20, 21, 22},
                    {30, 31, 32},
                    {40, 41, 42}};
    int *arr_d = 0;

    CUDA_SAFE_CALL(hipMallocPitch((void**)&arr_d,&pitch,n*sizeof(*arr_d),m));
    CUDA_SAFE_CALL(hipMemcpy2D(arr_d, pitch, arr, n*sizeof(arr[0][0]),
                                n*sizeof(arr[0][0]),m,hipMemcpyHostToDevice));
    tex.normalized = false;
    CUDA_SAFE_CALL (hipBindTexture2D (&tex_ofs, &tex, arr_d, &tex.channelDesc,
                                       n, m, pitch));
    if (tex_ofs !=0) {
        printf ("tex_ofs = %zu\n", tex_ofs);
        return EXIT_FAILURE;
    }
    printf ("reading texture:\n");
    kernel<<<1,1>>>(m, n);
    CHECK_LAUNCH_ERROR();
    CUDA_SAFE_CALL (hipDeviceSynchronize());
    return EXIT_SUCCESS;
}