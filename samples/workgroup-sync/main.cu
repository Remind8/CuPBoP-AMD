#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define NUM_BLOCKS 32
#define NUM_THREADS 256

__global__ void test_sync(int *output) {
    __shared__ int s[NUM_THREADS];
    __shared__ int placement;
    auto tIdx = threadIdx.x;
    if (tIdx == 0) {
        placement = blockIdx.x;
        s[placement] = 128;
    }

    __syncthreads();

    if (placement == blockIdx.x) {
        output[placement * blockDim.x + tIdx] = s[placement];
    } else {
        output[placement * blockDim.x + tIdx] = -2;
    }
}

int main() {
    int *output_d;
    int output_h[NUM_BLOCKS * NUM_THREADS];
    int num_inconsistent = 0;

    hipMalloc(&output_d, sizeof(int) * NUM_BLOCKS * NUM_THREADS);
    test_sync<<<NUM_BLOCKS, NUM_THREADS>>>(output_d);
    hipMemcpy(output_h, output_d, sizeof(int) * NUM_BLOCKS * NUM_THREADS, hipMemcpyDeviceToHost);

    for (int outputElement : output_h) {
        if (outputElement != 128) {
            num_inconsistent += 1;
            printf("got %d\n", outputElement);
        }
    }

    printf("%d inconsistent results\n", num_inconsistent);

    hipFree(output_d);
}
