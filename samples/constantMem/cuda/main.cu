

// #define BLOCKS 2

// __constant__ float factors[BLOCKS];

// __global__ void sum_and_multiply(const float * A, const float * B, float * C,
// const int size)
// {
//     int item = (blockIdx.x * blockDim.x) + threadIdx.x;
//     C[item] = (A[item] + B[item]) * factors[blockIdx.x];
// }

// http://codeofhonour.blogspot.com/2014/10/memories-from-cuda.html

#include <hip/hip_runtime.h>
#include <stdio.h>
// #include "helper_cuda.h"

#define S 4

/** HOST-side data */
int host_const[S] = {100, 200, 300, 400};

/* Some host-side vector */
int host_y[S] = {1, 2, 3, 4};

/* Address of device-side variable y */
int *dev_y;

/* Variable in the constant memory of the GPU */
__constant__ int dev_const_a[4];

/*
Output

101
202
303
404

*/

/**
 * Our kernel that adds the constant variable dev_const_a
 * to the given variable.
 */
__global__ void kernel(int *dev_var) {
    dev_var[threadIdx.x] += dev_const_a[threadIdx.x];
}

int main(void) {
    int data_size = S * sizeof(int);
    int i;

    /* Allocate space on the device for dev_y */
    hipMalloc((void **)&dev_y, data_size);

    /*
     * Copy data from the host to the device constant memory
     * host_const ---> dev_const_a
     */

    hipMemcpyToSymbol(HIP_SYMBOL(dev_const_a), host_const, data_size, 0,
                       hipMemcpyHostToDevice);

    /*
     * Copy data onto the device:
     * host_y ---> dev_y
     */

    hipMemcpy(dev_y, host_y, data_size, hipMemcpyHostToDevice);

    /*
     * Run the kernel
     */
    kernel<<<1, S>>>(dev_y);
    hipDeviceSynchronize();
    //   getLastCudaError("Kernel error");  /* Check for kernel errors! */

    /*
     * Copy data from the device back to the hsot
     *  dev_y ---> host_y
     */
    hipMemcpy(host_y, dev_y, data_size, hipMemcpyDeviceToHost);

    for (i = 0; i < S; i++) {
        printf("%d\n", host_y[i]);
    }
    if (dev_y != NULL)
        hipFree(dev_y);

    return 0;
}