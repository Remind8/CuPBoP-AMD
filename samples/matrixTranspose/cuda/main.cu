#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

// to compile:
// nvcc -O0 -o transpose transpose.cu -lm
// 
// to run:
// ./transpose 1024

// assume going forward 32x32 threads in each thread-block
#define BDIM 32

// reference "copy" kernel
__global__ void copy(int N, 
		     const float *  __restrict__ A,
		     float * __restrict__ AT){
	   
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  int idy = threadIdx.y + blockDim.y*blockIdx.y;

  // output
  if(idx<N && idy<N){
    AT[idx+idy*N] = A[idx+idy*N];
  }
}

// naive CUDA transpose kernel
__global__ void transposeV1(int N, 
			    const float * __restrict__ A, 
			    float * __restrict__ AT){
	   
  const int idx = threadIdx.x + blockDim.x*blockIdx.x;
  const int idy = threadIdx.y + blockDim.y*blockIdx.y;

  // output
  if(idx<N && idy<N){
    AT[idx+idy*N] = A[idy+idx*N]; // read A non-coalesced, write AT as coalesced
  }
}


// shared memory CUDA transpose kernel
__global__ void transposeV2(int N, 
			    const float *  __restrict__ A, 
			    float * __restrict__ AT){
	   
  const int idx = threadIdx.x + blockDim.x*blockIdx.x;
  const int idy = threadIdx.y + blockDim.y*blockIdx.y;

  __shared__ float s_A[BDIM][BDIM];

  // check this is a legal matrix entry
  if(idx<N && idy<N){
    s_A[threadIdx.y][threadIdx.x] = A[idx+idy*N]; // coalesced reads
  }

  // make sure all threads in this thread-block
  // have read into shared
  __syncthreads();

  // find coordinates of thread in transposed block
  const int idxT = threadIdx.x + blockDim.y*blockIdx.y;
  const int idyT = threadIdx.y + blockDim.x*blockIdx.x;

  // output
  if(idxT<N && idyT<N){
    AT[idxT+idyT*N] = s_A[threadIdx.x][threadIdx.y];
  }
}

// shared memory CUDA transpose kernel with padding to avoid smem bank conflicts
__global__ void transposeV3(int N, 
			    const float *  __restrict__ A, 
			    float * __restrict__ AT){
	   
  const int idx = threadIdx.x + blockDim.x*blockIdx.x;
  const int idy = threadIdx.y + blockDim.y*blockIdx.y;

  // pad by 1 to avoid 32-width bank-conflicts
  __shared__ float s_A[BDIM][BDIM+1];

  // check this is a legal matrix entry
  if(idx<N && idy<N){
    s_A[threadIdx.y][threadIdx.x] = A[idx+idy*N];
  }

  // ensure all threads in thread-block finish
  __syncthreads();

  // find coordinates of thread in transposed block
  const int idxT = threadIdx.x + blockDim.y*blockIdx.y;
  const int idyT = threadIdx.y + blockDim.x*blockIdx.x;

  // output
  if(idxT<N && idyT<N){
    AT[idxT+idyT*N] = s_A[threadIdx.x][threadIdx.y];    
  }
}

void validate(float *output, float *reference, int n) {
  int num_invalid = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      if (output[j+i*n] != reference[j+i*n]) {
        num_invalid += 1;
        fprintf(stderr, "output[%d, %d]: %d != reference: %d\n", j, i, output[j+i*n], reference[j+i*n]);
      }
    }
  }
  printf("%d inconsistencies\n", num_invalid);
}

int main(int argc, char **argv){
  
  int N = 2048;
  float *A  = (float*) calloc(N*N, sizeof(float));
  float *AT = (float*) calloc(N*N, sizeof(float));
  float *AT_ref = (float*) calloc(N*N, sizeof(float));

  printf("N=%d\n", N);

  for(int i=0;i<N;++i){
    for(int j=0;j<N;++j){
      A[j+i*N] = j;
      AT_ref[i+j*N] = j;
    }
  }

  float *c_A, *c_AT;
  size_t sz = N*N*sizeof(float); // size of matrix
  hipMalloc(&c_A, sz);
  hipMalloc(&c_AT, sz);
  hipMemcpy(c_A, A, sz, hipMemcpyHostToDevice);

  int Nblocks = (N+BDIM-1)/BDIM; // nearest Nblocks such that Nblocks * BDIM > N
  dim3 threadsPerBlock(BDIM,BDIM,1);
  dim3 blocks(Nblocks,Nblocks,1);

  printf("transposeV1");
  copy <<< blocks,threadsPerBlock >>> (N,c_A,c_AT);
  transposeV1 <<< blocks, threadsPerBlock >>> (N, c_A, c_AT);
  hipMemcpy(AT, c_AT, sz, hipMemcpyDeviceToHost);
  validate(AT, AT_ref, N);
  
  printf("transposeV2");
  transposeV2 <<< blocks, threadsPerBlock >>> (N, c_A, c_AT);
  hipMemcpy(AT, c_AT, sz, hipMemcpyDeviceToHost);
  validate(AT, AT_ref, N);

  printf("transposeV3");
  transposeV3 <<< blocks, threadsPerBlock >>> (N, c_A, c_AT);
  hipMemcpy(AT, c_AT, sz, hipMemcpyDeviceToHost);
  validate(AT, AT_ref, N);
  
  // --------------------------------------------------------------------------------

  hipError_t err = hipGetLastError();
  if(err != hipSuccess){
    fprintf(stderr, "CUDA ERROR: %s\n", 
	    hipGetErrorString(err));
  } else {
    fprintf(stderr, "No cuda error reported.\n");
  }

}