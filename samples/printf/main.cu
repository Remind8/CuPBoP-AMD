#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void test_printf()
{
    printf("%s (%d, %d, %d), blockId = (%d, %d, %d)\n", "threadId =", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z);
}

int main() {
    dim3 gridSize (2, 1, 2);
    dim3 blockSize (1, 2, 1);
    test_printf<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
}
