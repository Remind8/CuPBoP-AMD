#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

class TestDeviceClass {
  public:
    unsigned int value_a, value_b;

    __device__ TestDeviceClass() {
        value_a = 123;
        value_b = 321;
    }

    __device__ virtual unsigned get_value() { return value_a; }

    __device__ virtual unsigned get_value_fixed() { return value_a; }
};

class TestDeviceSubclass : public TestDeviceClass {
  public:
    __device__ virtual unsigned get_value() override { return value_b; }
};

__global__ void test_polymorphism() {
    auto test_a = TestDeviceClass();
    auto test_b = TestDeviceSubclass();
    TestDeviceClass *test_ref = &test_a;

    if ((threadIdx.x & 0x01) == 0) {
        test_ref = &test_b;
    }

    __syncthreads();
    printf("TID.x=%d, test_ref.get_value()=%d, test_ref.get_value_fixed()=%d\n",
           threadIdx.x, test_ref->get_value(), test_ref->get_value_fixed());
}

int main() {
    test_polymorphism<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
