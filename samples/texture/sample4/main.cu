#include "hip/hip_runtime.h"

// https://github.com/ArchaeaSoftware/cudahandbook/blob/master/texturing/tex1dfetch.cu

#include <stdio.h>
#include "hip/hip_runtime.h"
// #include <chError.h>

#define NUM_FLOATS 16

texture<float, 1, hipReadModeElementType> tex1;

__global__ void
TexReadout( float *out, size_t N )
{
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )   
    {
        out[i] = tex1Dfetch(tex1, i) * 0.5 ;
    }
}


__global__ void
TexReadout1( float *out, size_t N )
{
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )   
    {
        out[i] = tex1Dfetch(tex1, i) - 0.5 ;
    }
}


void
PrintTex( float *host, size_t N )
{
    float *device;
    hipError_t status;
    memset( host, 0, N*sizeof(float) );
    hipHostGetDevicePointer( (void **) &device, host, 0 );
    
    TexReadout1<<<2,384>>>( device, N );
    hipDeviceSynchronize();
    for (int i = 0; i < N; i++) {
        printf( "%.2f ", host[i] );
    }
    printf( "\n" );
    // Output
    // 0.00 1.00 2.00 3.00 4.00 5.00 6.00 7.00 8.00 9.00 10.00 11.00 12.00 13.00 14.00 15.00
Error:;
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    float *p = 0;
    float *finHost;
    float *finDevice;

    float *foutHost;
    float *foutDevice;
    hipError_t status;
    hipDeviceProp_t props;

    hipSetDeviceFlags(hipDeviceMapHost);
    hipGetDeviceProperties( &props, 0);
    hipMalloc( (void **) &p, NUM_FLOATS*sizeof(float));
    hipHostAlloc( (void **) &finHost, NUM_FLOATS*sizeof(float), hipHostMallocMapped);
    hipHostGetDevicePointer( (void **) &finDevice, finHost, 0 );

    hipHostAlloc( (void **) &foutHost, NUM_FLOATS*sizeof(float), hipHostMallocMapped);
    hipHostGetDevicePointer( (void **) &foutDevice, foutHost, 0 );

    for ( int i = 0; i < NUM_FLOATS; i++ ) {
        finHost[i] = (float) i;
    }

    {
        size_t offset;
        hipBindTexture( &offset, tex1, finDevice, NUM_FLOATS*sizeof(float));
    }

    PrintTex( foutHost, NUM_FLOATS );

    ret = 0;
Error:
    hipFree( p );
    hipHostFree( finHost );
    hipHostFree( foutHost );
    return ret;
}