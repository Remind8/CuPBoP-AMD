#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "stdio.h"
// #include "hip/hip_vector_types.h"

// will need export CPATH=~/hip-11.5/samples/common/inc:$CPATH

// https://forums.developer.nvidia.com/t/tex1dfetch-apparently-returning-incorrect-value/38795
// https://github.com/ricsonc/linear_vs_texture_memory_hip/blob/master/main.cu

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef float4 typ;
typedef uint4 typ3;

// typedef float2 typ1;
// typedef float3 typ2;
// typedef int2 typ4;
// typedef int3 typ5;


texture<typ, 1> texref;
// texture<typ3, 1> texref2;
// texture<typ3, 2> texref2;


// texture<typ1, 1> texref1;
// texture<typ2, 1> texref2;
// texture<typ3, 1> texref3;
// texture<typ4, 1> texref4;
// texture<typ5, 1> texref5;




// texture<typ1, 2> texref5;
// texture<typ, 3> texref6;

/* 1D texture memory */

__global__ void touch1Dtexture(void* outPtr_, long M){
    long N = gridDim.x * blockDim.x;
    long i = blockIdx.x * blockDim.x + threadIdx.x;

    typ* outPtr = (typ*) outPtr_;

    for(; i < M-2; i += N) {
        float4 a = tex1Dfetch(texref, i);
        // float4 b = tex1Dfetch(texref, i+1); 
        // float4 c = tex1Dfetch(texref, i+2);  
        // float4 d = tex1Dfetch(texref, i+3);
        outPtr[i] = (
           a 
        );
        //  if (i < 10) {
        //     printf("%d: %f ,", i, a.x);
        // }
    }
}


// __global__ void touch1Dtexture1(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long i = blockIdx.x * blockDim.x + threadIdx.x;

//     typ1* outPtr = (typ1*) outPtr_;

//     for(; i < M-2; i += N) {
//         float2 a = tex1Dfetch(texref1, i);
//         // float4 b = tex1Dfetch(texref, i+1); 
//         // float4 c = tex1Dfetch(texref, i+2);  
//         // float4 d = tex1Dfetch(texref, i+3);
//         outPtr[i] = (
//            a 
//         );
//     }
// }

// __global__ void touch1Dtexture2(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long i = blockIdx.x * blockDim.x + threadIdx.x;

//     typ2* outPtr = (typ2*) outPtr_;

//     for(; i < M-2; i += N) {
//         float3 a = tex1Dfetch(texref2, i);
//         // float4 b = tex1Dfetch(texref, i+1); 
//         // float4 c = tex1Dfetch(texref, i+2);  
//         // float4 d = tex1Dfetch(texref, i+3);
//         outPtr[i] = (
//            a 
//         );
//     }
// }

// __global__ void touch1Dtexture3(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long i = blockIdx.x * blockDim.x + threadIdx.x;

//     typ3* outPtr = (typ3*) outPtr_;

//     for(; i < M-2; i += N) {
//         int4 a = tex1Dfetch(texref3, i);
//         // float4 b = tex1Dfetch(texref, i+1); 
//         // float4 c = tex1Dfetch(texref, i+2);  
//         // float4 d = tex1Dfetch(texref, i+3);
//         outPtr[i] = (
//            a 
//         );
//     }
// }

// __global__ void touch1Dtexture4(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long i = blockIdx.x * blockDim.x + threadIdx.x;

//     typ4* outPtr = (typ4*) outPtr_;

//     for(; i < M-2; i += N) {
//         int2 a = tex1Dfetch(texref4, i);
//         // float4 b = tex1Dfetch(texref, i+1); 
//         // float4 c = tex1Dfetch(texref, i+2);  
//         // float4 d = tex1Dfetch(texref, i+3);
//         outPtr[i] = (
//            a 
//         );
//     }
// }

// __global__ void touch1Dtexture5(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long i = blockIdx.x * blockDim.x + threadIdx.x;

//     typ5* outPtr = (typ5*) outPtr_;

//     for(; i < M-2; i += N) {
//         int3 a = tex1Dfetch(texref5, i);
//         // float4 b = tex1Dfetch(texref, i+1); 
//         // float4 c = tex1Dfetch(texref, i+2);  
//         // float4 d = tex1Dfetch(texref, i+3);
//         outPtr[i] = (
//            a 
//         );
//     }
// }



int time1Dtexture()
{
    void* refPtr;
    void* outPtr;
    long M = 1000L*1000L*100L;
    int blocks = 65536;
    int threads = 64;

    gpuCheck( hipMalloc(&refPtr, M*sizeof(typ)) );
    gpuCheck( hipMalloc(&outPtr, M*sizeof(typ)) );
     size_t offset;
    gpuCheck( hipBindTexture(&offset, texref, refPtr, M) );
    
    touch1Dtexture<<<blocks, threads>>>(outPtr, M);
    
    gpuCheck( hipPeekAtLastError() );
    gpuCheck( hipDeviceSynchronize() );


    gpuCheck( hipUnbindTexture(texref) );    
    gpuCheck( hipFree(refPtr) );
    gpuCheck( hipFree(outPtr) );
    
}

/* 2D linear memory */

// __global__ void touch2Dtexture(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long ix = blockIdx.x * blockDim.x + threadIdx.x;
//     long iy = blockIdx.y * blockDim.y + threadIdx.y;

//     typ1* outPtr = (typ1*) outPtr_;

//     for(; ix < M-1; ix += N) {
//         for(; iy < M-1; iy += N) {
//             outPtr[ix*M+iy] = (
//                 tex2D(texref2, ix, iy) +
//                 tex2D(texref2, ix+1, iy) +
//                 tex2D(texref2, ix, iy+1) +
//                 tex2D(texref2, ix+1, iy+1)
//             );
//         }
//     }
// }

// /* 3D texture memory */

// __global__ void touch3Dtexture(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long ix = blockIdx.x * blockDim.x + threadIdx.x;
//     long iy = blockIdx.y * blockDim.y + threadIdx.y;
//     long iz = blockIdx.z * blockDim.z + threadIdx.z;

//     typ* outPtr = (typ*) outPtr_;

//     for(; ix < M-1; ix += N) {
//         for(; iy < M-1; iy += N) {
//             for(; iz < M-1; iz += N) {
//                 outPtr[ix*M*M+iy*M+iz] = (
//                     tex3D(texref3, ix, iy, iz) +                    
//                     tex3D(texref3, ix, iy, iz+1) +
//                     tex3D(texref3, ix, iy+1, iz) +
//                     tex3D(texref3, ix+1, iy, iz)
//                 );
//             }
//         }
//     }
// }

// void time2Dtexture()
// {
//     long M = 10000;
//     dim3 blocks(256, 256);
//     dim3 threads(8,8);

//     void* outPtr;
//     gpuCheck( hipMalloc(&outPtr, M*M*sizeof(typ)) );

//     hipArray *refPtr;
//     gpuCheck( hipMallocArray(&refPtr, &texref2.channelDesc, M, M) );
//     gpuCheck( hipBindTextureToArray(texref2, refPtr) );
    
   
//     touch2Dtexture<<<blocks, threads>>>(outPtr, M);
    
    
//     gpuCheck( hipPeekAtLastError() );
//     gpuCheck( hipDeviceSynchronize() );



//     gpuCheck( hipUnbindTexture(texref) );    
//     gpuCheck( hipFreeArray(refPtr) );
//     gpuCheck( hipFree(outPtr) );
    
// }


// void time3Dtexture()
// {
//     unsigned long M = 465;
//     dim3 blocks(32,32,32);
//     dim3 threads(4,4,4);

//     void* outPtr;
//     gpuCheck( hipMalloc(&outPtr, M*M*M*sizeof(typ)) );
    
//     hipArray* refPtr;
//     gpuCheck( hipMalloc3DArray(&refPtr, &texref2.channelDesc, {M, M, M}) );
//     gpuCheck( hipBindTextureToArray(texref3, refPtr) );
    
   
//     touch3Dtexture<<<blocks, threads>>>(outPtr, M);
    
    
//     gpuCheck( hipPeekAtLastError() );
//     gpuCheck( hipDeviceSynchronize() );


//     gpuCheck( hipUnbindTexture(texref) );    
//     gpuCheck( hipFreeArray(refPtr) );
//     gpuCheck( hipFree(outPtr) );
    
// }

int main() {
    time1Dtexture();
}