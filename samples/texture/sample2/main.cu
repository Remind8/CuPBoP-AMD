#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "stdio.h"
#include "samples/common/inc/hip/hip_vector_types.h"

// will need export CPATH=~/cuda-11.5/samples/common/inc:$CPATH

// https://forums.developer.nvidia.com/t/tex1dfetch-apparently-returning-incorrect-value/38795
// https://github.com/ricsonc/linear_vs_texture_memory_cuda/blob/master/main.cu

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef float4 typ;

texture<typ, 1> texref;
texture<typ, 2> texref2;
texture<typ, 3> texref3;

/* 1D texture memory */

__global__ void touch1Dtexture(void* outPtr_, long M){
    long N = gridDim.x * blockDim.x;
    long i = blockIdx.x * blockDim.x + threadIdx.x;

    typ* outPtr = (typ*) outPtr_;

    for(; i < M-2; i += N) {
        float4 a = tex1Dfetch(texref, i);
        // float4 b = tex1Dfetch(texref, i+1); 
        // float4 c = tex1Dfetch(texref, i+2);  
        // float4 d = tex1Dfetch(texref, i+3);
        outPtr[i] = (
           a 
        );
        if (i < 10) {
            printf("%d: %f ,", i, a.x);
        }
    }
}

void time1Dtexture()
{
    void* refPtr;
    void* outPtr;
    long M = 1000L*1000L*100L;
    int blocks = 65536;
    int threads = 64;

    gpuCheck( hipMalloc(&refPtr, M*sizeof(typ)) );
    gpuCheck( hipMalloc(&outPtr, M*sizeof(typ)) );
    size_t offset;
    gpuCheck( hipBindTexture(&offset, texref, refPtr, M) );
    
    printf("Kernel Launch \n");
    touch1Dtexture<<<blocks, threads>>>(outPtr, M);
    
    gpuCheck( hipPeekAtLastError() );
    gpuCheck( hipDeviceSynchronize() );


    gpuCheck( hipUnbindTexture(texref) );    
    gpuCheck( hipFree(refPtr) );
    gpuCheck( hipFree(outPtr) );
    
}

/* 2D linear memory */

// __global__ void touch2Dtexture(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long ix = blockIdx.x * blockDim.x + threadIdx.x;
//     long iy = blockIdx.y * blockDim.y + threadIdx.y;

//     typ* outPtr = (typ*) outPtr_;

//     for(; ix < M-1; ix += N) {
//         for(; iy < M-1; iy += N) {
//             outPtr[ix*M+iy] = (
//                 tex2D(texref2, ix, iy) +
//                 tex2D(texref2, ix+1, iy) +
//                 tex2D(texref2, ix, iy+1) +
//                 tex2D(texref2, ix+1, iy+1)
//             );
//         }
//     }
// }

// /* 3D texture memory */

// __global__ void touch3Dtexture(void* outPtr_, long M){
//     long N = gridDim.x * blockDim.x;
//     long ix = blockIdx.x * blockDim.x + threadIdx.x;
//     long iy = blockIdx.y * blockDim.y + threadIdx.y;
//     long iz = blockIdx.z * blockDim.z + threadIdx.z;

//     typ* outPtr = (typ*) outPtr_;

//     for(; ix < M-1; ix += N) {
//         for(; iy < M-1; iy += N) {
//             for(; iz < M-1; iz += N) {
//                 outPtr[ix*M*M+iy*M+iz] = (
//                     tex3D(texref3, ix, iy, iz) +                    
//                     tex3D(texref3, ix, iy, iz+1) +
//                     tex3D(texref3, ix, iy+1, iz) +
//                     tex3D(texref3, ix+1, iy, iz)
//                 );
//             }
//         }
//     }
// }

// void time2Dtexture()
// {
//     long M = 10000;
//     dim3 blocks(256, 256);
//     dim3 threads(8,8);

//     void* outPtr;
//     gpuCheck( hipMalloc(&outPtr, M*M*sizeof(typ)) );

//     hipArray *refPtr;
//     gpuCheck( hipMallocArray(&refPtr, &texref2.channelDesc, M, M) );
//     gpuCheck( hipBindTextureToArray(texref2, refPtr) );
    
   
//     touch2Dtexture<<<blocks, threads>>>(outPtr, M);
    
    
//     gpuCheck( hipPeekAtLastError() );
//     gpuCheck( hipDeviceSynchronize() );



//     gpuCheck( hipUnbindTexture(texref) );    
//     gpuCheck( hipFreeArray(refPtr) );
//     gpuCheck( hipFree(outPtr) );
    
// }


// void time3Dtexture()
// {
//     unsigned long M = 465;
//     dim3 blocks(32,32,32);
//     dim3 threads(4,4,4);

//     void* outPtr;
//     gpuCheck( hipMalloc(&outPtr, M*M*M*sizeof(typ)) );
    
//     hipArray* refPtr;
//     gpuCheck( hipMalloc3DArray(&refPtr, &texref2.channelDesc, {M, M, M}) );
//     gpuCheck( hipBindTextureToArray(texref3, refPtr) );
    
   
//     touch3Dtexture<<<blocks, threads>>>(outPtr, M);
    
    
//     gpuCheck( hipPeekAtLastError() );
//     gpuCheck( hipDeviceSynchronize() );


//     gpuCheck( hipUnbindTexture(texref) );    
//     gpuCheck( hipFreeArray(refPtr) );
//     gpuCheck( hipFree(outPtr) );
    
// }

int main() {
    time1Dtexture();
}