#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void empty(int *N) {
    int a = 5;
    *N = 10;
    int myNum[3] = {10, 20, 30};
    int al = sizeof(myNum) / sizeof(myNum[0]);
    for (int i = 0; i < al; i++) {
        *N = *N + myNum[i] * a;
    }
}

struct myStruct {
    int a;
    int b;
    int c;
};

__global__ void empty_byval(myStruct x, int *N) {
    *N = x.a + x.b + x.c;
}

// 10 + 10 * 5 = 60
// 60 + 6- * 5
// __global__ void empty(int* N) {
//   int a = 5;
//   *N = 10;
//   int myNum[3] = {10, 20, 30};
//   int al = sizeof(myNum)/sizeof(myNum[0]);
//   for(int i = 0; i< al; i++) {
//     *N = *N + myNum[i] * a;
//   }
// }

int main(int argc, char **argv) {
    int *host_n;
    int *device_n;
    myStruct x { 3, 2, 1 };

    host_n = (int *)malloc(sizeof(int));
    hipMalloc(&device_n, sizeof(int));

    // With simple argument
    empty<<<1, 1>>>(device_n);
    hipMemcpy(host_n, device_n, sizeof(int), hipMemcpyDeviceToHost);

    printf("Value: %d\n", *host_n); // 310
    if (*host_n == 310) {
        printf("Success 1\n");
    }

    // With one byval() struct
    empty_byval<<<1, 1>>>(x, device_n);
    hipMemcpy(host_n, device_n, sizeof(int), hipMemcpyDeviceToHost);

    printf("Value: %d\n", *host_n);
    if (*host_n == 6) {
        printf("Success 2\n");
    }

    hipFree(device_n);
    return 0;
}
