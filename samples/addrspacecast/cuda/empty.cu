#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void empty(int* N) {
  int a = 5;
  *N = 10;
  int myNum[3] = {10, 20, 30};
  int al = sizeof(myNum)/sizeof(myNum[0]);
  for(int i = 0; i< al; i++) {
    *N = *N + myNum[i] * a; 
  }
}
// 10 + 10 * 5 = 60 
// 60 + 6- * 5
// __global__ void empty(int* N) {
//   int a = 5;
//   *N = 10;
//   int myNum[3] = {10, 20, 30};
//   int al = sizeof(myNum)/sizeof(myNum[0]);
//   for(int i = 0; i< al; i++) {
//     *N = *N + myNum[i] * a;
//   }
// }



int main(int argc, char **argv) { 
  int * host_n;
  int * device_n;

  host_n = (int*)malloc(sizeof(int));
  hipMalloc(&device_n, sizeof(int));

  empty<<<1,1>>>(device_n);

  hipMemcpy(host_n, device_n, sizeof(int), hipMemcpyDeviceToHost);

  printf("\n value: %d \n", *host_n); // 310
  if(*host_n == 310) {
    printf(" Success ");
  }

  hipFree(device_n);
  return 0;
}
