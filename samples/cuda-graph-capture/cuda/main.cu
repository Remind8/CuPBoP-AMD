#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 4096
#define BLOCK_SIZE 32
#define GRID_SIZE (N / BLOCK_SIZE)
#define NUM_LAUNCHES 4096

__global__ void kernel_a(int *a, int *b, int *output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    output[i] = i * a[i] + b[i];
}

__global__ void kernel_b(int *a, int *b, int *output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    output[i] = a[i] + i * b[i];
}

__global__ void kernel_c(int *out_a, int *out_b, int *output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    output[i] = out_a[i] + out_b[i];
}

__host__ void prepare_inputs(int *a, int *b, int *d_a, int *d_b) {
    for (int i = 0; i < N; ++i) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);
}

__host__ void verify_outputs(int *a, int *b, int *output) {
    for (int i = 0; i < N; ++i) {
        int expected = (i + 1) * (a[i] + b[i]);
        if (output[i] != expected) {
            printf("\nError at index %d: expected %d, got %d (a = %d, b = %d)", i, expected, output[i], a[i], b[i]);
        }
    }
}

int main() {
    int *a, *b, *out_a, *out_b, *output;
    int *d_a, *d_b, *d_out_a, *d_out_b, *d_output;
    time_t t;
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    fprintf(stdout, "sizeof(cudaStream_t)=%lu, sizeof(cudaGraph_t)=%lu, sizeof(cudaGraphExec_t)=%lu\n", sizeof(hipStream_t), sizeof(hipGraph_t), sizeof(hipGraphExec_t));
    srand((unsigned) time(&t));

    // Allocate memory
    a = (int *) malloc(N * sizeof(int));
    b = (int *) malloc(N * sizeof(int));
    out_a = (int *) malloc(N * sizeof(int));
    out_b = (int *) malloc(N * sizeof(int));
    output = (int *) malloc(N * sizeof(int));

    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_out_a, N * sizeof(int));
    hipMalloc((void **) &d_out_b, N * sizeof(int));
    hipMalloc((void **) &d_output, N * sizeof(int));

    hipStreamCreate(&stream);

    // Capture graph - a simple graph where
    // A   B
    //  \ /
    //   C
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    kernel_a<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(d_a, d_b, d_out_a);
    kernel_b<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(d_a, d_b, d_out_b);
    kernel_c<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(d_out_a, d_out_b, d_output);
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    // Execute graphs multiple times
    for (int i = 0; i < NUM_LAUNCHES; ++i) {
        fprintf(stdout, "\rLaunching instance %d/%d with %d inputs...", i + 1, NUM_LAUNCHES, N);
        fflush(stdout);
        prepare_inputs(a, b, d_a, d_b);
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
        hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);
        verify_outputs(a, b, output);
    }
    fprintf(stdout, "\n");

    hipStreamDestroy(stream);
    
    // Free memory
    free(a);
    free(b);
    free(out_a);
    free(out_b);
    free(output);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out_a);
    hipFree(d_out_b);
    hipFree(d_output);

    return 0;
}
