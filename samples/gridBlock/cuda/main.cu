#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void test_blockdim(int *x, int *y, int *z)
{
    *x = blockDim.x;
    *y = blockDim.y;
    *z = blockDim.z;
}

__global__ void test_griddim(int *x, int *y, int *z)
{
    *x = gridDim.x;
    *y = gridDim.y;
    *z = gridDim.z;
}

int main()
{
    dim3 grid_size (1, 4, 5);
    dim3 block_size (3, 8, 2);

    int hostbuf[6];
    int *host_ntidx = hostbuf,
        *host_ntidy = hostbuf + 1,
        *host_ntidz = hostbuf + 2,
        *host_nctaidx = hostbuf + 3,
        *host_nctaidy = hostbuf + 4,
        *host_nctaidz = hostbuf + 5;
    int *device_ntidx, *device_ntidy, *device_ntidz, *device_nctaidx, *device_nctaidy, *device_nctaidz;

    hipMalloc(&device_ntidx, sizeof(int));
    hipMalloc(&device_ntidy, sizeof(int));
    hipMalloc(&device_ntidz, sizeof(int));
    hipMalloc(&device_nctaidx, sizeof(int));
    hipMalloc(&device_nctaidy, sizeof(int));
    hipMalloc(&device_nctaidz, sizeof(int));

    test_blockdim<<<grid_size, block_size>>>(device_ntidx, device_ntidy, device_ntidz);
    test_griddim<<<grid_size, block_size>>>(device_nctaidx, device_nctaidy, device_nctaidz);

    hipDeviceSynchronize();
    hipMemcpy(host_ntidx, device_ntidx, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_ntidy, device_ntidy, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_ntidz, device_ntidz, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_nctaidx, device_nctaidx, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_nctaidy, device_nctaidy, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_nctaidz, device_nctaidz, sizeof(int), hipMemcpyDeviceToHost);

    if (*host_ntidx != block_size.x) {
        printf("Error: host_ntid.x = %d != %d\n", *host_ntidx, block_size.x);
        return 1;
    }

    if (*host_ntidy != block_size.y) {
        printf("Error: host_ntid.y = %d != %d\n", *host_ntidy, block_size.y);
        return 1;
    }

    if (*host_ntidz != block_size.z) {
        printf("Error: host_ntid.z = %d != %d\n", *host_ntidz, block_size.z);
        return 1;
    }

    if (*host_nctaidx != grid_size.x) {
        printf("Error: host_nctaid.x = %d != %d\n", *host_nctaidx, grid_size.x);
        return 1;
    }

    if (*host_nctaidy != grid_size.y) {
        printf("Error: host_nctaid.y = %d != %d\n", *host_nctaidy, grid_size.y);
        return 1;
    }

    if (*host_nctaidz != grid_size.z) {
        printf("Error: host_nctaid.z = %d != %d\n", *host_nctaidz, grid_size.z);
        return 1;
    }

    printf("Success!\n");

    hipFree(device_ntidx);
    hipFree(device_ntidy);
    hipFree(device_ntidz);
    hipFree(device_nctaidx);
    hipFree(device_nctaidy);
    hipFree(device_nctaidz);

    return 0;
}
