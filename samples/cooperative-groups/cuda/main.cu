#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <numeric>

namespace cg = cooperative_groups;

__global__ void test_thread_block() {
    auto g = cg::this_thread_block();

    int i = g.thread_rank();
    int s = g.size();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d\n", i, s, threadIdx.x, blockIdx.x);
    }
}

__global__ void test_grid() {
    auto g = cg::this_grid();

    int i = g.thread_rank();
    int s = g.size();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d\n", i, s, threadIdx.x, blockIdx.x);
    }
}

int main(void) {
    test_thread_block<<<4, 32>>>();
    hipDeviceSynchronize();

    void *kernel_args[] = {};
    hipLaunchCooperativeKernel((void*) test_grid, 4, 32, kernel_args);
    hipDeviceSynchronize();
}
