#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <numeric>

namespace cg = cooperative_groups;

__device__ int reduce_sum(cg::thread_group g, int *temp,
                          int val) {

    int lane = g.thread_rank();

    for (int i = g.size() / 2; i > 0; i /= 2) {

        temp[lane] = val;
        g.sync();
        if (lane < i)
            val += temp[lane + i];
        g.sync();
    }
    // printf("val returned in reduced_sum =%d\n",val);
    return val;
}

__device__ int thread_sum(int *input, int n) {
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n / 4;
         i += blockDim.x * gridDim.x) {
        int4 in = ((int4 *)input)[i];
        sum += in.x + in.y + in.z + in.w;
    }
    // printf("sum returned from thrad_sum =%d\n",sum);
    return sum;
}

__global__ void sum_kernel_block(int *sum, int *input, int n) {
    extern __shared__ int temp[];
    int my_sum = thread_sum(input, n);
    auto g = cg::this_thread_block();
    int block_sum = reduce_sum(g, temp, my_sum);

    if (g.thread_rank() == 0)
        atomicAdd(sum, block_sum);
}

int main(void) {
    int n = 1 << 24;
    int blockSize = 256;
    int nBlocks = (n + blockSize - 1) / blockSize;
    int sharedBytes = blockSize * sizeof(int);
    int result;
    int *sum, *data;
    
    hipMallocManaged(&sum, sizeof(int));
    hipMallocManaged(&data, n * sizeof(int));

    std::fill_n(data, n, rand());
    int cpu_res = std::accumulate(data, data + n, 0);

    hipMemset(sum, 0, sizeof(int));

    sum_kernel_block<<<nBlocks, blockSize, sharedBytes>>>(sum, data, n);
    hipMemcpy(&result, sum, sizeof(int), hipMemcpyDeviceToHost);
    printf("Sum of 16M array number = %d, expecting %d\n", result, cpu_res);

    hipFree(sum);
    hipFree(data);
}
