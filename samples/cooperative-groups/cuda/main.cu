#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>

namespace cg = cooperative_groups;

__global__ void test_thread_block() {
    auto g = cg::this_thread_block();

    int i = g.thread_rank();
    int s = g.size();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("[device] Rank %d, Size %d, TID.x = %d, BID.x = %d, dim=(%d, %d, %d)\n",
               i, s, threadIdx.x, blockIdx.x, dim.x, dim.y, dim.z);
    }
}

__global__ void test_grid() {
    auto g = cg::this_grid();

    int i = g.thread_rank();
    int s = g.size();
    bool v = g.is_valid();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("[device] Rank %d, Size %d, TID.x = %d, BID.x = %d, valid = %d, dim=(%d, "
               "%d, %d)\n",
               i, s, threadIdx.x, blockIdx.x, v, dim.x, dim.y, dim.z);
    }
}

__global__ void test_block_tiling() {
    auto g = cg::this_thread_block();

    g.sync();

    cg::thread_block_tile<4> tile4 = cg::tiled_partition<4>(g);
    auto mgrank = tile4.size();

    tile4.sync();
    if (tile4.meta_group_rank() == 0) {
        printf("[device] tid=<(%d, %d, %d), (%d, %d, %d)>, t16.mgr=%d, t16.mgs=%d, "
               "t16.size=%d, t16.rank=%d\n",
               threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
               blockIdx.z, tile4.meta_group_rank(), tile4.meta_group_size(),
               tile4.size(), tile4.thread_rank());
    }
}

__global__ void test_coalesced_threads() {
    auto blockGroup = cg::this_thread_block();
    if ((blockGroup.thread_rank() % 3) == 0) {
        auto coalescedGroup = cg::coalesced_threads();
        auto rank = coalescedGroup.thread_rank();
        auto size = coalescedGroup.size();
        coalescedGroup.sync();
        printf("[device] Rank %d, Size %d, TID.x = %d, BID.x = %d\n", rank, size,
               threadIdx.x, blockIdx.x);
    }
}

int main(void) {
    printf("[host] Running test_coalesced_threads\n");
    test_coalesced_threads<<<1, 64>>>();
    hipDeviceSynchronize();

    printf("[host] Running test_thread_block\n");
    test_thread_block<<<4, 32>>>();
    hipDeviceSynchronize();

    printf("[host] Running test_grid\n");
    test_block_tiling<<<4, 32>>>();
    hipDeviceSynchronize();

    // TODO: future kernel launches seems to hang after this??
    void *kernel_args[] = {};
    printf("[host] Running test_grid\n");
    hipLaunchCooperativeKernel((void *)test_grid, 4, 32, kernel_args);
    hipDeviceSynchronize();

    return 0;
}
