#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>

namespace cg = cooperative_groups;

__global__ void test_thread_block() {
    auto g = cg::this_thread_block();

    int i = g.thread_rank();
    int s = g.size();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d, dim=(%d, %d, %d)\n",
               i, s, threadIdx.x, blockIdx.x, dim.x, dim.y, dim.z);
    }
}

__global__ void test_grid() {
    auto g = cg::this_grid();

    int i = g.thread_rank();
    int s = g.size();
    bool v = g.is_valid();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d, valid = %d, dim=(%d, "
               "%d, %d)\n",
               i, s, threadIdx.x, blockIdx.x, v, dim.x, dim.y, dim.z);
    }
}

__global__ void test_block_tiling() {
    auto g = cg::this_thread_block();

    g.sync();

    cg::thread_block_tile<4> tile4 = cg::tiled_partition<4>(g);
    auto mgrank = tile4.size();
    
    tile4.sync();
    if (tile4.meta_group_rank() == 0) {
        printf("tid=<(%d, %d, %d), (%d, %d, %d)>, t16.mgr=%d, t16.mgs=%d, "
               "t16.size=%d, t16.rank=%d\n",
               threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
               blockIdx.z, tile4.meta_group_rank(), tile4.meta_group_size(),
               tile4.size(), tile4.thread_rank());
    }
}

int main(void) {
    test_thread_block<<<4, 32>>>();
    hipDeviceSynchronize();

    test_block_tiling<<<4, 32>>>();
    hipDeviceSynchronize();

    // TODO: future kernel launches seems to hang after this??
    // void *kernel_args[] = {};
    // hipLaunchCooperativeKernel((void *)test_grid, 4, 32, kernel_args);
    // hipDeviceSynchronize();

    return 0;
}
