#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>

namespace cg = cooperative_groups;

__global__ void test_thread_block() {
    auto g = cg::this_thread_block();

    int i = g.thread_rank();
    int s = g.size();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d, dim=(%d, %d, %d)\n", i, s, threadIdx.x,
               blockIdx.x, dim.x, dim.y, dim.z);
    }
}

__global__ void test_grid() {
    auto g = cg::this_grid();

    int i = g.thread_rank();
    int s = g.size();
    bool v = g.is_valid();
    auto dim = g.group_dim();

    g.sync();

    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d, valid = %d, dim=(%d, %d, %d)\n", i, s,
               threadIdx.x, blockIdx.x, v, dim.x, dim.y, dim.z);
    }
}

int main(void) {
    test_thread_block<<<4, 32>>>();
    hipDeviceSynchronize();

    void *kernel_args[] = {};
    hipLaunchCooperativeKernel((void *)test_grid, 4, 32, kernel_args);
    hipDeviceSynchronize();
}
