#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <numeric>

namespace cg = cooperative_groups;

// __device__ int reduce_sum(cg::thread_group g, int *temp,
//                           int val) {

//     int lane = g.thread_rank();

//     for (int i = g.size() / 2; i > 0; i /= 2) {

//         temp[lane] = val;
//         g.sync();
//         if (lane < i)
//             val += temp[lane + i];
//         g.sync();
//     }
//     // printf("val returned in reduced_sum =%d\n",val);
//     return val;
// }

// __device__ int thread_sum(int *input, int n) {
//     int sum = 0;
//     for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n / 4;
//          i += blockDim.x * gridDim.x) {
//         int4 in = ((int4 *)input)[i];
//         sum += in.x + in.y + in.z + in.w;
//     }
//     // printf("sum returned from thrad_sum =%d\n",sum);
//     return sum;
// }

// struct tg_data {
//     unsigned int is_tiled : 1;
//     unsigned int type : 7;
//     unsigned int size : 24;
//     // packed to 4b
//     unsigned int metaGroupSize : 16;
//     unsigned int metaGroupRank : 16;
//     // packed to 8b
//     unsigned int mask;
//     // packed to 12b
//     unsigned int _res;
// };

__global__ void sum_kernel_block(int *sum, int *input, int n) {
    // extern __shared__ int temp[];
    // int my_sum = thread_sum(input, n);
    auto g = cg::this_thread_block();
    // int block_sum = reduce_sum(g, temp, my_sum);
    int i = g.thread_rank();
    int s = g.size();

    g.sync();
    
    if (threadIdx.x < 5 && blockIdx.x == 1) {
        printf("Rank %d, Size %d, TID.x = %d, BID.x = %d\n", i, s, threadIdx.x, blockIdx.x);
    }

    // if (threadIdx.x == 0 && blockIdx.x == 0) {
    //     printf("Size is %d, rank is %d\n", g.size(), g.thread_rank());
    // }

    // if (g.thread_rank() == 0)
    //     atomicAdd(sum, block_sum);
}

int main(void) {
    int n = 1 << 24;
    int blockSize = 256;
    int nBlocks = (n + blockSize - 1) / blockSize;
    int sharedBytes = blockSize * sizeof(int);
    int result;
    int *sum, *data;
    
    hipMallocManaged(&sum, sizeof(int));
    hipMallocManaged(&data, n * sizeof(int));

    std::fill_n(data, n, rand());
    int cpu_res = std::accumulate(data, data + n, 0);

    hipMemset(sum, 0, sizeof(int));

    sum_kernel_block<<<nBlocks, blockSize, sharedBytes>>>(sum, data, n);
    hipMemcpy(&result, sum, sizeof(int), hipMemcpyDeviceToHost);
    printf("Sum of 16M array number = %d, expecting %d\n", result, cpu_res);

    hipFree(sum);
    hipFree(data);
}
