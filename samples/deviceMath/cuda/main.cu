#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"
#include <cmath>

#define NUM_TEST_CALLS 2

// __device__ float wrap_sin(float input) {
//     return sin(input);
// }

// __device__ float wrap_cos(float input) {
//     return cos(input);
// }

#define TEST_SEQ(in, out) \
    out[0] = sin(in[0]); \
    out[1] = cos(in[1]);

void test_math_host(float *in, float *out) {
    TEST_SEQ(in, out)
}

__global__ void test_math(float *in, float *out) {
    TEST_SEQ(in, out)
}

void check_is_close(int cs, float result, float ref, float error = 1e-6) {
    if (fabs(result - ref) > error) {
        fprintf(stderr, "test case %d failed: %f != %f\n", cs, result, ref);
    } else {
        printf("test case %d passed\n", cs);
    }
}

int main() {
    float in_h[NUM_TEST_CALLS];
    float *in_d;
    float out_h[NUM_TEST_CALLS];
    float out_exp_h[NUM_TEST_CALLS];
    float *out_d;

    hipMalloc(&in_d, NUM_TEST_CALLS * sizeof(float));
    hipMalloc(&out_d, NUM_TEST_CALLS * sizeof(float));

    in_h[0] = 1.123;
    in_h[1] = 2.321;
    
    hipMemcpy(in_d, in_h, NUM_TEST_CALLS * sizeof(float), hipMemcpyHostToDevice);
    test_math_host(in_h, out_exp_h);
    test_math<<<1, 1>>>(in_d, out_d);
    hipMemcpy(out_h, out_d, NUM_TEST_CALLS * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < NUM_TEST_CALLS; ++i) {
        check_is_close(i, out_h[i], out_exp_h[i]);
    }

    hipFree(in_d);
    hipFree(out_d);
}
