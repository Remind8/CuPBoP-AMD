#include <hip/hip_runtime.h>

#include "stdio.h"
#include "math.h"
#include <cmath>

#define NUM_TEST_CALLS 26

#define TEST_SEQ(in, out, counter) \
    out[counter] = (int) ::lgamma(10.0f); counter += 1; \
    out[counter] = (int) ::exp(1.0f); counter += 1; \
    out[counter] = (int) ::log(1.0f); counter += 1; \
    out[counter] = (int) ::log10(1000.0f); counter += 1; \
    out[counter] = (int) ::log1p(0.0f); counter += 1; \
    out[counter] = (int) ::log2(1000.0f); counter += 1; \
    out[counter] = (int) ::expm1(1.0f); counter += 1; \
    out[counter] = (int) ::cos(0.0f); counter += 1; \
    out[counter] = (int) ::sin(0.0f); counter += 1; \
    out[counter] = (int) ::sqrt(100.0f); counter += 1; \
    out[counter] = (int) ::ceil(2.4f); counter += 1; \
    out[counter] = (int) ::floor(2.7f); counter += 1; \
    out[counter] = (int) ::trunc(2.7f); counter += 1; \
    out[counter] = (int) ::cosh(1.0f); counter += 1; \
    out[counter] = (int) ::acosh(1.0f); counter += 1; \
    out[counter] = (int) ::asin(1.0f); counter += 1; \
    out[counter] = (int) ::sinh(1.0f); counter += 1; \
    out[counter] = (int) ::asinh(1.0f); counter += 1; \
    out[counter] = (int) ::tan(0.0f); counter += 1; \
    out[counter] = (int) ::atan(1.0f); counter += 1; \
    out[counter] = (int) ::tanh(1.0f); counter += 1; \
    out[counter] = (int) ::erf(10.0f); counter += 1; \
    out[counter] = (int) ::erfc(10.0f); counter += 1; \
    out[counter] = (int) ::round(2.3f); counter += 1; \
    out[counter] = (int) ::pow(2.0f, 10.0f); counter += 1; \
    out[counter] = (int) ::atan2(7.0f, 0.0f); counter += 1; \
    /* out[counter] = (int) ::isnan(0.0f); counter += 1; */ \
    /* out[counter] = (int) ::isinf(0.0f); counter += 1; */

void test_math_host(float *in, float *out) {
    int counter = 0;
    TEST_SEQ(in, out, counter)
}

__global__ void test_math(float *in, float *out) {
    int counter = 0;
    TEST_SEQ(in, out, counter)
}

void check_is_close(int cs, float result, float ref, float error = 1e-6) {
    if (fabs(result - ref) > error) {
        fprintf(stderr, "test case %d failed: %f != %f\n", cs, result, ref);
    } else {
        printf("test case %d passed\n", cs);
    }
}

int main() {
    float in_h[NUM_TEST_CALLS];
    float *in_d;
    float out_h[NUM_TEST_CALLS];
    float out_exp_h[NUM_TEST_CALLS];
    float *out_d;

    hipMalloc(&in_d, NUM_TEST_CALLS * sizeof(float));
    hipMalloc(&out_d, NUM_TEST_CALLS * sizeof(float));

    in_h[0] = 1.123;
    in_h[1] = 2.321;
    
    hipMemcpy(in_d, in_h, NUM_TEST_CALLS * sizeof(float), hipMemcpyHostToDevice);
    test_math_host(in_h, out_exp_h);
    test_math<<<1, 1>>>(in_d, out_d);
    hipMemcpy(out_h, out_d, NUM_TEST_CALLS * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < NUM_TEST_CALLS; ++i) {
        check_is_close(i, out_h[i], out_exp_h[i]);
    }

    hipFree(in_d);
    hipFree(out_d);
}
